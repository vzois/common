#include "hip/hip_runtime.h"
#include "CudaHelper.h"

/*
 * Setup kernel for random number generator
 *
 */
__global__ void setup_kernel()
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets different seed, a different sequence
	number, no offset */
	hiprand_init(7 + id, id, 0, &devStates[id]);
}

/*
 * Initializing random State in GPU
 *
 */
__host__ void init_rand_gpu(){
	setup_kernel<<< RAND_BLOCKS, RAND_THREADS >>>();
	error = hipDeviceSynchronize();
	handleDeviceErrors(error, "Error Initializing Rand GPU in CudaHelper");
}

/*
 * Device error handling
 *
 */
__host__ void handleDeviceErrors(hipError_t error, std::string comment){
	if (error != hipSuccess){ std::cout << "Cuda Error: " << comment << "," << hipGetErrorString(error) << std::endl; }
}

/*
 * Allocating device memory
 * addr: Address in GPU
 * size: Data size in bytes
 * msg: Error message to be displayed
 */
template void allocDevMem<unsigned int>(unsigned int **addr, unsigned int size, std::string msg);
template void allocDevMem<unsigned short>(unsigned short **addr, unsigned int size, std::string msg);
template void allocDevMem<int>(int **addr, unsigned int size, std::string msg);
template void allocDevMem<short>(short **addr, unsigned int size, std::string msg);
template void allocDevMem<bool>(bool **addr, unsigned size, std::string msg);
template void allocDevMem<float>(float **addr, unsigned size, std::string msg);
template void allocDevMem<double>(double **addr, unsigned size, std::string msg);

template<class V>
__host__ void allocDevMem(V **addr, unsigned int size, std::string msg){
	error = hipMalloc(&(*addr), size);
	handleDeviceErrors(error, "Error Allocating device " + msg);
}

/*
 * Allocating Pinned Memory
 *
 * addr: Address in GPU
 * size: Data size in bytes
 * msg: error message to be displayed
 */
template void allocHostMem<unsigned int>(unsigned int **addr, unsigned int size, std::string msg);
template void allocHostMem<unsigned short>(unsigned short **addr, unsigned int size, std::string msg);
template void allocHostMem<int>(int **addr, unsigned int size, std::string msg);
template void allocHostMem<short>(short **addr, unsigned int size, std::string msg);
template void allocHostMem<bool>(bool **addr, unsigned int size, std::string msg);
template void allocHostMem<float>(float **addr, unsigned int size, std::string msg);
template void allocHostMem<double>(double **addr, unsigned int size, std::string msg);


template<class V>
__host__ void allocHostMem(V **addr, unsigned int size, std::string msg){
	error = hipHostMalloc(&(*addr), size);
	handleDeviceErrors(error, "Error Allocating host "+msg);
}

/*
 * Copy Arrays to Device
 * to: GPU address
 * from: DRAM address
 * size: data size in bytes
 * msg: error message to be displayed
 */

template void safeCpyToDevice<unsigned int>(unsigned int *to, unsigned int *from, unsigned int size, std::string msg);
template void safeCpyToDevice<unsigned short>(unsigned short *to, unsigned short *from, unsigned int size, std::string msg);
template void safeCpyToDevice<int>(int *to, int *from, unsigned int size, std::string msg);
template void safeCpyToDevice<short>(short *to, short *from, unsigned int size, std::string msg);
template void safeCpyToDevice<bool>(bool *to, bool *from, unsigned int size, std::string msg);
template void safeCpyToDevice<float>(float *to, float *from, unsigned int size, std::string msg);
template void safeCpyToDevice<double>(double *to, double *from, unsigned int size, std::string msg);

template<class V>
__host__ void safeCpyToDevice(V *to, V *from, unsigned int size, std::string msg){
	error = hipMemcpy(to,from,size,hipMemcpyHostToDevice);
	handleDeviceErrors(error, "Error Copying to device "+ msg);
}

template void safeCpyToHost<unsigned int>(unsigned int *to, unsigned int *from, unsigned int size, std::string msg);
template void safeCpyToHost<unsigned short>(unsigned short *to, unsigned short *from, unsigned int size, std::string msg);
template void safeCpyToHost<int>(int *to, int *from, unsigned int size, std::string msg);
template void safeCpyToHost<short>(short *to, short *from, unsigned int size, std::string msg);
template void safeCpyToHost<bool>(bool *to, bool *from, unsigned int size, std::string msg);
template void safeCpyToHost<float>(float *to, float *from, unsigned int size, std::string msg);
template void safeCpyToHost<double>(double *to, double *from, unsigned int size, std::string msg);

template<class V>
__host__ void safeCpyToHost(V *to, V *from, unsigned int size, std::string msg){
	error = hipMemcpy(to, from, size, hipMemcpyDeviceToHost);
	handleDeviceErrors(error, "Error Copying to device " + msg);
}


/*
 * Copying to symbol
 *
 */
template void safeCpyToSymbol<unsigned int>(unsigned int *symbol, unsigned int *data, std::string msg);
template void safeCpyToSymbol<unsigned short>(unsigned short *symbol, unsigned short *data, std::string msg);

template<class V>
__host__ void safeCpyToSymbol(V *symbol, V *data, std::string msg){
	unsigned int k = 13;
	error = hipMemcpyToSymbol(HIP_SYMBOL(symbol), &k, sizeof(V));
	handleDeviceErrors(error, "Error Copying symbol "+ msg);
}

/*
 * Print current Device Specs
 *
 */
__host__ hipError_t printDeviceSpecs(bool print){
	hipDeviceProp_t prop;
	hipError_t error = hipSuccess;
	int devs = 0;
	
	error = hipGetDeviceCount(&devs);
	if (!print) return error;
	if (error != hipSuccess){ handleDeviceErrors(error, "Error Getting Number of Devices");  return error; }
	std::cout << std::endl;
	std::cout << "Number of Devices: (" << devs << ")" << std::endl;

	for (int i = 0; i < devs; i++){
		error = hipGetDeviceProperties(&prop, i);
		if (error != hipSuccess){ handleDeviceErrors(error, "Error Reading Device Properties");  return error; }
		std::cout << "<<<<<< Device " << i << " >>>>>>" << std::endl;

		std::cout << "Device Name: " << prop.name << std::endl;

		std::cout << "Device Compute Mode: " << prop.computeMode <<std::endl;
		std::cout << "Device Major Compute Capability: " << prop.major << std::endl;
		std::cout << "Device Minor Compute Capability: " << prop.minor << std::endl;

		std::cout << "Number of AsyncEngineCount: " << prop.asyncEngineCount << std::endl;
		std::cout << "Global Memory Size: " << prop.totalGlobalMem << std::endl;
		std::cout << "Constant Memory Size: " << prop.totalConstMem << std::endl;

		std::cout << "Number of Multiprocessors: " << prop.multiProcessorCount << std::endl;
		std::cout << "Shared Memory Per Multiprocessor: " << prop.sharedMemPerMultiprocessor << std::endl;
		std::cout << "Shared Memory Per Block: " << ((float)prop.sharedMemPerMultiprocessor) << std::endl;
		
		/*int x = 0;
		error = hipDeviceGetAttribute(&x, hipDeviceAttributeMaxBlockDimX, 0);
		std::cout << "Device Block Number X:" << x << endl;
		error = hipDeviceGetAttribute(&x, hipDeviceAttributeMaxBlockDimY, 0);
		std::cout << "Device Block Number Y:" << x << endl;
		error = hipDeviceGetAttribute(&x, hipDeviceAttributeMaxBlockDimZ, 0);
		std::cout << "Device Block Number Z:" << x << endl;*/

		std::cout << "Maximum Grid Size (X,Y,Z): (" << prop.maxGridSize[0] << "),("
			<< prop.maxGridSize[1] << "),(" << prop.maxGridSize[2] << ")" << std::endl;

		std::cout << "Maximum Threads Per Block: " << prop.maxThreadsPerBlock<< std::endl;
		std::cout << "Maximum Number of Blocks (X,Y,Z): (" << prop.maxThreadsDim[0] << "),("
			<< prop.maxThreadsDim[1] << "),(" << prop.maxThreadsDim[2] << ")" << std::endl;

	}
	std::cout << std::endl;

	return hipSuccess;
}

dim3 grid_1D(unsigned int N, unsigned int data_per_block){
	return dim3((N - 1) / data_per_block + 1, 1, 1);
}

//AMPLIFY = # ELEMENTS PER THREAD
dim3 grid_1D(unsigned int N, unsigned int data_per_block, unsigned int amplification){
	return dim3((N - 1) / (data_per_block*amplification) + 1, 1, 1);
}

dim3 block_1D(unsigned int data_per_block){
	return dim3(data_per_block, 1, 1);
}

void print_grid(dim3 grid, dim3 block){
	std::cout<<"grid("<<grid.x <<","<<grid.y << "," << grid.z <<")"<<std::endl;
	std::cout<<"block("<<block.x <<","<<block.y << "," << block.z <<")"<<std::endl;
}
